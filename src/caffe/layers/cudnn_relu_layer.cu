#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Forward_gpu(bottom, top);
  }

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(hipdnnActivationForward(this->handle_,
        HIPDNN_ACTIVATION_RELU,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
}

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.rel