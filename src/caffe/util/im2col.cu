#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_index = index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * kernel_h * kernel_w;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im_ptr[i * width + j] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename Dtype>
void im2col_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    Dtype* data_col) {
  // We are going to launch chan